#include "gpu_t.cuh"

class gpus_t {
    std::vector<gpu_t*> gpus;
public:
    gpus_t()
    {
        int n;
        if (hipGetDeviceCount(&n) != hipSuccess)
            return;
        for (int id = 0; id < n; id++) {
            hipDeviceProp_t prop;
            if (hipGetDeviceProperties(&prop, id) == hipSuccess &&
                prop.major >= 7) {
                hipSetDevice(id);
                gpus.push_back(new gpu_t(id, prop));
            }
        }
        hipSetDevice(0);
    }
    ~gpus_t()
    {   for (auto* ptr: gpus) delete ptr;   }

    inline gpu_t* operator[](size_t i) const
    {   return gpus[i];   }
    inline size_t ngpus() const
    {   return gpus.size();   }

    static gpus_t& all()
    {
        static gpus_t all_gpus;
        return all_gpus;
    }
};

gpu_t& select_gpu(int id)
{
    hipSetDevice(id);
    return *gpus_t::all()[id];
}

size_t ngpus()
{   return gpus_t::all().ngpus();   }

extern "C" bool cuda_available() { return gpus_t::all().ngpus() != 0; }
